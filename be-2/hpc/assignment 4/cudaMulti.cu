
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;
__global__ void matmul(int* A, int* B, int* C, int N) {
    int Row = blockIdx.y*blockDim.y+threadIdx.y;
    int Col = blockIdx.x*blockDim.x+threadIdx.x;
    if (Row < N && Col < N) {
        int Pvalue = 0;
        for (int k = 0; k < N; k++) {
            Pvalue += A[Row*N+k] * B[k*N+Col];
        }
        C[Row*N+Col] = Pvalue;
    }
}

int main() {
    int N;
    cout <<"enter size of matrix n: ";
    cin >> N;
    int size = N * N * sizeof(int);
    int* A, * B, * C;
    int* dev_A, * dev_B, * dev_C;
    hipHostMalloc(&A, size, hipHostMallocDefault);
    hipHostMalloc(&B, size, hipHostMallocDefault);
    hipHostMalloc(&C, size, hipHostMallocDefault);
    hipMalloc(&dev_A, size);
    hipMalloc(&dev_B, size);
    hipMalloc(&dev_C, size);

    // Initialize matrices A and B
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            A[i*N+j] = i*N+j;
            B[i*N+j] = j*N+i;
        }
    }

    hipMemcpy(dev_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_B, B, size, hipMemcpyHostToDevice);

    dim3 dimBlock(N, N);
    dim3 dimGrid(N/dimBlock.x, N/dimBlock.y);

    matmul<<<dimGrid, dimBlock>>>(dev_A, dev_B, dev_C, N);

    hipMemcpy(C, dev_C, size, hipMemcpyDeviceToHost);

    
    cout << "\n1st matrix" << endl;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            cout << A[i*N+j] << " "; 
        }
        std::cout << std::endl;
    }
    cout << "\n2nd matrix" << endl;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            cout << B[i*N+j] << " "; 
        }
        std::cout << std::endl;
    }

    // Print the result
    cout << "\n resultant matrix" << endl;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            cout << C[i*N+j] << " ";
        }
        std::cout << std::endl;
    }

    // Free memory
    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_C);
    hipHostFree(A);
    hipHostFree(B);
    hipHostFree(C);

    return 0;
}
