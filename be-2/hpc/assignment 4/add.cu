#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>

const int VECTOR_SIZE = 100;

__global__ void vectorAdd(const int* a, const int* b, int* c, int size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size)
    {
        c[tid] = a[tid] + b[tid];
    }
}

int main()
{
    // Allocate memory for host vectors
    int* h_a = new int[VECTOR_SIZE];
    int* h_b = new int[VECTOR_SIZE];
    int* h_c = new int[VECTOR_SIZE];

    // Initialize host vectors with random values
    for (int i = 0; i < VECTOR_SIZE; ++i)
    {
        h_a[i] = std::rand() % 100;
        h_b[i] = std::rand() % 100;
    }

    // Allocate memory for device vectors
    int* d_a, * d_b, * d_c;
    hipMalloc(&d_a, VECTOR_SIZE * sizeof(int));
    hipMalloc(&d_b, VECTOR_SIZE * sizeof(int));
    hipMalloc(&d_c, VECTOR_SIZE * sizeof(int));

    // Copy input data from host to device
    hipMemcpy(d_a, h_a, VECTOR_SIZE * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, VECTOR_SIZE * sizeof(int), hipMemcpyHostToDevice);

    // Launch the vector addition kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (VECTOR_SIZE + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, VECTOR_SIZE);

    // Copy the result from device to host
    hipMemcpy(h_c, d_c, VECTOR_SIZE * sizeof(int), hipMemcpyDeviceToHost);

   
    std::cout << "\nfirst vector: \n";
    for(int i=0; i<VECTOR_SIZE; i++)
    {
        std::cout << h_a[i] << " ";
    }
    std::cout << "\nsecond vector: \n";
    for(int i=0; i<VECTOR_SIZE; i++)
    {
        std::cout << h_b[i] << " ";
    }
     // Print the first few elements of the resulting vector
    std::cout << "\nfirst few elements of the resulting vector: \n";
    for (int i = 0; i < VECTOR_SIZE; ++i)
    {
        std::cout << h_c[i] << " ";
        
    }
    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Free host memory
    delete[] h_a;
    delete[] h_b;
    delete[] h_c;

    return 0;
}
